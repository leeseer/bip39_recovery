
#include <hip/hip_runtime.h>
// kernel.cu
extern "C" __global__ void recover_kernel(
    void* tasks_ptr,
    unsigned int tasks_len,
    void* target_ptr,
    void* result_ptr
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < tasks_len) {
        ((unsigned char*)result_ptr)[idx] = 0;
    }
}